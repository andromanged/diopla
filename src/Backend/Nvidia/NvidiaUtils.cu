// Copyright (c) 2019, Zpalmtree
//
// Please see the included LICENSE file for more information.

#include <iostream>
#include <map>
#include <vector>

#include "Nvidia/Argon2.h"
#include "Utilities/ColouredMsg.h"

std::vector<std::tuple<std::string, bool, int>> getNvidiaDevicesActual()
{
    std::vector<std::tuple<std::string, bool, int>> devices;

    int numberDevices;
    hipGetDeviceCount(&numberDevices);

    for (int i = 0; i < numberDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        devices.push_back(std::make_tuple(prop.name, true, i));
    }

    return devices;
}

void printNvidiaHeader()
{
    std::cout << InformationMsg<std::string>("* ") << WhiteMsg<std::string>("NVIDIA DEVICES", 25);

    int numberDevices;
    hipGetDeviceCount(&numberDevices);

    if (numberDevices == 0)
    {
        std::cout << WarningMsg<std::string>("None found") << std::endl;
        return;
    }

    std::map<std::string, std::vector<hipDeviceProp_t>> gpus;

    for (int i = 0; i < numberDevices; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        std::string deviceName = prop.name;

        gpus[deviceName].push_back(prop);
    }

    int i = 0;

    for (const auto gpu : gpus)
    {
        const auto name = gpu.first;
        const auto properties = gpu.second;

        if (i != 0)
        {
            std::cout << ", ";
        }

        size_t numDevices = properties.size();

        std::cout << SuccessMsg<size_t>(numDevices) << SuccessMsg<std::string>("x ") << SuccessMsg<std::string>(name);

        i++;
    }

    std::cout << std::endl;
}

uint32_t getNoncesPerRun(const size_t scratchpadSize, const uint32_t gpuIndex)
{
    hipDeviceProp_t properties;

    /* Figure out how much memory we have available */
    hipGetDeviceProperties(&properties, gpuIndex);

    const size_t ONE_MB = 1024 * 1024;
    const size_t ONE_GB = ONE_MB * 1024;

    size_t memoryAvailable = (properties.totalGlobalMem / ONE_GB - 1) * (ONE_GB / ONE_MB);

    /* The amount of nonces we're going to try per kernel launch */
    uint32_t noncesPerRun = (memoryAvailable * ONE_MB) / (1024 * scratchpadSize);
    noncesPerRun = (noncesPerRun / 128) * 128;

    return noncesPerRun;
}
